#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <string>

#include "hip/hip_runtime.h"


typedef unsigned size_type;

const size_type MOD = 17;
const size_type ROOT = 3;
const size_type ROOT_ORDER = 1 << 4;

// бинарное возведение в степень по модулю (a^n(mod))
size_type binary_pow(size_type a, size_type n, size_type mod) {
    size_type res = 1;
    while (n) {
        if (n & 1) res = res * a % mod;
        a = a * a % mod;
        n >>= 1;
    }
    return res;
}

// находит обратный элемент как n^(mod-2)
size_type reverse(size_type n, size_type mod) {
    return binary_pow(n, mod - 2, mod);
}

void print_vector(const std::string& name, const size_type *vec, size_type size, bool polynomial) {
    std::cout << name << ": ";
    for (auto i = 0; i < size; i++) {
        if (polynomial) {
            if (vec[i] != 0)
                std::cout << vec[i] << "x^" << i << " ";
        } else {
            if (i != size - 1)
                std::cout << vec[i] << ", ";
            else
                std::cout << vec[i];
        }
    }
    std::cout << std::endl;
}

void check_error(hipError_t err) {
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "error code: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void fft(std::vector<size_type>& a, bool invert) {
    int n = (int)a.size();

    for (int i = 1, j = 0; i < n; ++i) {
        int bit = n >> 1;
        for (; j >= bit; bit >>= 1)
            j -= bit;
        j += bit;
        if (i < j)
            std::swap(a[i], a[j]);
    }

    for (int len = 2; len <= n; len <<= 1) {
        int wlen = invert ? reverse(ROOT, MOD) : ROOT;
        for (int i = len; i < ROOT_ORDER; i <<= 1)
            wlen = int(wlen * 1ll * wlen % MOD);
        for (int i = 0; i < n; i += len) {
            int w = 1;
            for (int j = 0; j < len / 2; ++j) {
                int u = a[i + j], v = int(a[i + j + len / 2] * 1ll * w % MOD);
                a[i + j] = u + v < MOD ? u + v : u + v - MOD;
                a[i + j + len / 2] = u - v >= 0 ? u - v : u - v + MOD;
                w = int(w * 1ll * wlen % MOD);
            }
        }
    }

    if (invert) {
        int nrev = reverse(n, MOD);
        for (int i = 0; i < n; ++i)
            a[i] = int(a[i] * 1ll * nrev % MOD);
    }
}

__global__ void bit_reversal(size_type *vec, size_type size, size_type sizeLog2) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        auto bitsNumber = sizeof(i) * 8;
        size_type reverse_i = 0;
        for (auto j = 0; j < bitsNumber; j++)
            if ((i & (1 << j)))
                reverse_i |= 1 << ((bitsNumber - 1) - j);
        reverse_i >>= (bitsNumber - sizeLog2);
        if (i < reverse_i) {
            size_type temp = vec[i];
            vec[i] = vec[reverse_i];
            vec[reverse_i] = temp;
        }
    }
}

__global__ void fft_butterflies(size_type *vec, size_type size, size_type len, size_type wlen) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size && i % len == 0; i += blockDim.x * gridDim.x) {
        int w = 1;
        for (int j = 0; j < len / 2; ++j) {
            int u = vec[i + j], v = int(vec[i + j + len / 2] * 1ll * w % MOD);
            vec[i + j] = u + v < MOD ? u + v : u + v - MOD;
            vec[i + j + len / 2] = u - v >= 0 ? u - v : u - v + MOD;
            w = int(w * 1ll * wlen % MOD);
        }
    }
}

__global__ void invert_fft_result(size_type *vec, size_type size, size_type nrev) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        vec[i] = int(vec[i] * 1ll * nrev % MOD);
    }
}

void parallel_fft(size_type *vec, size_type size, bool invert, int numBlocks, int blockSize) {
    hipError_t err = hipSuccess;

    bit_reversal <<<numBlocks, blockSize>>>(vec, size, log2(size));

    hipDeviceSynchronize();
    check_error(err);

    for (int len = 2; len <= size; len <<= 1) {
        int wlen = invert ? reverse(ROOT, MOD) : ROOT;
        for (int i = len; i < ROOT_ORDER; i <<= 1)
            wlen = int(wlen * 1ll * wlen % MOD);

        fft_butterflies<<<numBlocks, blockSize>>>(vec, size, len, wlen);
    }

    hipDeviceSynchronize();
    check_error(err);

    if (invert) {
        int nrev = reverse(size, MOD);
        invert_fft_result<<<numBlocks, blockSize>>>(vec, size, nrev);
        hipDeviceSynchronize();
        check_error(err);
    }
}

__global__ void multiply_vectors(size_type *vec1, size_type *vec2, size_type *res_vec) {
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalIdx < ROOT_ORDER)
        res_vec[globalIdx] = vec1[globalIdx] * vec2[globalIdx] % MOD;
}

int main() {
    hipError_t err = hipSuccess;
    int blockSize = 512;
    size_type numBlocks = (ROOT_ORDER + blockSize - 1) / blockSize;

    bool random = true;
    std::cout << "random values - 1, hardcoded values - 0: ";
    std::cin >> random;
    std::cout << std::endl;

    size_type *vector1, *vector2;
    hipMallocManaged(&vector1, ROOT_ORDER * sizeof(size_type));
    hipMallocManaged(&vector2, ROOT_ORDER * sizeof(size_type));
    if (random) {
        for (auto i = 0; i < ROOT_ORDER / 2; i++) {
            vector1[i] = rand() % MOD;
            vector2[i] = rand() % MOD;
        }
    } else {
        std::vector<size_type> test_vec1 = { 7, 8, 3, 4 };
        std::vector<size_type> test_vec2 = { 9, 5, 16 };
        test_vec1.resize(ROOT_ORDER);
        test_vec2.resize(ROOT_ORDER);
        std::copy(test_vec1.begin(), test_vec1.end(), vector1);
        std::copy(test_vec2.begin(), test_vec2.end(), vector2);
    }

    print_vector("vector1", vector1, ROOT_ORDER, true);
    print_vector("vector2", vector2, ROOT_ORDER, true);

    parallel_fft(vector1, ROOT_ORDER, false, numBlocks, blockSize);
    parallel_fft(vector2, ROOT_ORDER, false, numBlocks, blockSize);

    print_vector("parallel FFT vector1", vector1, ROOT_ORDER, false);
    print_vector("parallel FFT vector2", vector2, ROOT_ORDER, false);

    size_type *res_vec;
    hipMallocManaged(&res_vec, ROOT_ORDER * sizeof(size_type));
    multiply_vectors<<<numBlocks, blockSize>>>(vector1, vector2, res_vec);
    hipDeviceSynchronize();
    check_error(err);

    hipFree(vector1);
    hipFree(vector2);
    check_error(err);

    print_vector("multiplied FFT vector", res_vec, ROOT_ORDER, false);

    parallel_fft(res_vec, ROOT_ORDER, true, numBlocks, blockSize);

    print_vector("result", res_vec, ROOT_ORDER, true);

    hipFree(res_vec);
    check_error(err);
}
